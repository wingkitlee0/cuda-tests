#include <cstdio>
#include <cstdlib>
#include <vector>
#include <memory>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ float generate(hiprandState* globalState, int ind)
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void addToCount(int N, int *y, hiprandState* globalState)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
while (id < N)
{
    int number = generate(globalState, id) * 1000000;
    printf("%i\n", number);

    atomicAdd(&(y[0]), number);
    id += blockDim.x * gridDim.x;
}
}

int main(int argc, char** argv)
{
  int N = 20;
  int *d_y;
  //int *y, *d_y;
  //y = (int*)malloc(N*sizeof(int));
  std::vector<int> y(N);

  hipMalloc(&d_y, N * sizeof(int));
  hipMemcpy(d_y, &y[0], N * sizeof(int), hipMemcpyHostToDevice);
  //cudaMemcpy(d_y, y, N * sizeof(int), cudaMemcpyHostToDevice);

  hiprandState* devStates;
  hipMalloc (&devStates, N * sizeof(hiprandState));
  srand(time(0));
  /** ADD THESE TWO LINES **/
  int seed = rand();
  setup_kernel<<<2, 100>>>(devStates,seed);
  /** END ADDITION **/
  addToCount<<<2, 100>>>(N, d_y, devStates);

  hipMemcpy(&y[0], d_y, N*sizeof(int), hipMemcpyDeviceToHost);
  //cudaMemcpy(y, d_y, N*sizeof(int), cudaMemcpyDeviceToHost);

  printf("final = %i\n", y[0]);
  
}
