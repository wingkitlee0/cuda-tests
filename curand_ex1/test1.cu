#include <cstdio>
#include <cstdlib>
#include <vector>
#include <memory>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ float generate(hiprandState* globalState, int ind)
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void addToCount(int N, int *y, hiprandState* globalState)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
while (id < N)
{
    int number = generate(globalState, id) * 1000000;
    printf("%i\n", number);

    atomicAdd(&(y[0]), number);
    id += blockDim.x * gridDim.x;
}
}

int main(void)
{
  int N = 10;
  int *y, *d_y;
  y = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_y, N * sizeof(int));
  hipMemcpy(d_y, y, N * sizeof(int), hipMemcpyHostToDevice);

  hiprandState* devStates;
  hipMalloc (&devStates, N * sizeof(hiprandState));
  srand(time(0));
  /** ADD THESE TWO LINES **/
  int seed = rand();
  setup_kernel<<<2, N>>>(devStates,seed);
  /** END ADDITION **/
  addToCount<<<2, N>>>(N, d_y, devStates);

  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);
  printf("final = %i\n", *y);
}
